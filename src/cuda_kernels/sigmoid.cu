
#include <hip/hip_runtime.h>
extern "C"   
{
    __global__ void sigmoid(const int lengthA, const double *a,  double *b)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i<lengthA)
	{
	  b[i] = exp(-a[i]);  
	  b[i] = 1.0/(1.0+b[i]);
	}
    }
}